#include "hip/hip_runtime.h"
/*
nvcc main.cu -o main -lopencv_core -lopencv_imgcodecs -lopencv_highgui -lopencv_imgproc
*/

#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

typedef struct {
    float r, g, b;
} Color;

// Kernel per assegnare ciascun pixel al cluster più vicino
__global__ void assign_pixels_to_centroids(Color* pixels, Color* centroids, int* assignments, int num_pixels, int num_centroids) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_pixels) {
        float min_dist = FLT_MAX;
        int closest_centroid = 0;
        Color pixel = pixels[idx];

        // Calcolo della distanza tra il pixel e ogni centroide
        for (int i = 0; i < num_centroids; i++) {
            float dist = (pixel.r - centroids[i].r) * (pixel.r - centroids[i].r) +
                         (pixel.g - centroids[i].g) * (pixel.g - centroids[i].g) +
                         (pixel.b - centroids[i].b) * (pixel.b - centroids[i].b);
            if (dist < min_dist) {
                min_dist = dist;
                closest_centroid = i;
            }
        }
        assignments[idx] = closest_centroid;
    }
}

// Kernel per aggiornare i centroidi senza atomiche
__global__ void update_centroids(Color* pixels, int* assignments, Color* centroids, int* cluster_sizes, int num_pixels, int num_centroids) {
    __shared__ float shared_centroids_r[256];
    __shared__ float shared_centroids_g[256];
    __shared__ float shared_centroids_b[256];
    __shared__ int shared_cluster_sizes[256];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x < num_centroids) {
        shared_centroids_r[threadIdx.x] = 0.0f;
        shared_centroids_g[threadIdx.x] = 0.0f;
        shared_centroids_b[threadIdx.x] = 0.0f;
        shared_cluster_sizes[threadIdx.x] = 0;
    }
    __syncthreads();

    // Aggiorna i centroidi in base ai pixel assegnati (memoria condivisa)
    if (idx < num_pixels) {
        int cluster_idx = assignments[idx];
        atomicAdd(&shared_centroids_r[cluster_idx], pixels[idx].r);
        atomicAdd(&shared_centroids_g[cluster_idx], pixels[idx].g);
        atomicAdd(&shared_centroids_b[cluster_idx], pixels[idx].b);
        atomicAdd(&shared_cluster_sizes[cluster_idx], 1);
    }
    __syncthreads();

    // Ogni thread calcola il centroide finale
    if (threadIdx.x < num_centroids) {
        if (shared_cluster_sizes[threadIdx.x] > 0) {
            centroids[threadIdx.x].r = shared_centroids_r[threadIdx.x] / shared_cluster_sizes[threadIdx.x];
            centroids[threadIdx.x].g = shared_centroids_g[threadIdx.x] / shared_cluster_sizes[threadIdx.x];
            centroids[threadIdx.x].b = shared_centroids_b[threadIdx.x] / shared_cluster_sizes[threadIdx.x];
        }
    }
}

// Kernel per creare l'immagine cartoonizzata
__global__ void create_cartoon_image(Color* pixels, int* assignments, Color* centroids, int num_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_pixels) {
        Color pixel = pixels[idx];
        int closest_centroid = assignments[idx];

        // Assegna il pixel al centroide più vicino
        pixel.r = centroids[closest_centroid].r;
        pixel.g = centroids[closest_centroid].g;
        pixel.b = centroids[closest_centroid].b;

        pixels[idx] = pixel;
    }
}

int main() {
    const int num_clusters = 90;  // Numero di colori finali (clusters)
    const int max_iterations = 50;

    // Carica l'immagine utilizzando OpenCV
    cv::Mat image = cv::imread("images/image.jpg");

    if (image.empty()) {
        printf("Errore nel caricare l'immagine\n");
        return -1;
    }

    int width = image.cols;
    int height = image.rows;
    int num_pixels = width * height;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Converti l'immagine in un array di pixel RGB
    Color* pixels = (Color*)malloc(num_pixels * sizeof(Color));
    for (int i = 0; i < num_pixels; i++) {
        cv::Vec3b color = image.at<cv::Vec3b>(i / width, i % width);
        pixels[i].b = (float)color[0] / 255.0f;
        pixels[i].g = (float)color[1] / 255.0f;
        pixels[i].r = (float)color[2] / 255.0f;
    }

    srand(time(NULL));

    // Centroidi iniziali (in GPU)
    Color* centroids = (Color*)malloc(num_clusters * sizeof(Color));
    for (int i = 0; i < num_clusters; i++) {
        centroids[i].r = (float)rand() / RAND_MAX;
        centroids[i].g = (float)rand() / RAND_MAX;
        centroids[i].b = (float)rand() / RAND_MAX;
    }

    printf("R: %f\n", centroids[15].r);
    printf("G: %f\n", centroids[15].g);
    printf("B: %f\n", centroids[15].b);

    Color* d_pixels;
    Color* d_centroids;
    int* d_assignments;
    int* d_cluster_sizes;

    // Allocazione della memoria su GPU
    hipMalloc(&d_pixels, num_pixels * sizeof(Color));
    hipMalloc(&d_centroids, num_clusters * sizeof(Color));
    hipMalloc(&d_assignments, num_pixels * sizeof(int));
    hipMalloc(&d_cluster_sizes, num_clusters * sizeof(int));

    hipMemcpy(d_pixels, pixels, num_pixels * sizeof(Color), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, num_clusters * sizeof(Color), hipMemcpyHostToDevice);

    // Inizializza cluster_sizes su GPU
    int* cluster_sizes = (int*)malloc(num_clusters * sizeof(int));
    memset(cluster_sizes, 0, num_clusters * sizeof(int));
    hipMemcpy(d_cluster_sizes, cluster_sizes, num_clusters * sizeof(int), hipMemcpyHostToDevice);

    // Esegui K-means per un numero di iterazioni
    for (int iter = 0; iter < max_iterations; iter++) {
        // Passo 1: Assegna i pixel ai centri
        assign_pixels_to_centroids<<<(num_pixels + 255) / 256, 256>>>(d_pixels, d_centroids, d_assignments, num_pixels, num_clusters);
        hipDeviceSynchronize();

        // Passo 2: Aggiorna i centroidi
        update_centroids<<<(num_clusters + 255) / 256, 256>>>(d_pixels, d_assignments, d_centroids, d_cluster_sizes, num_pixels, num_clusters);
        hipDeviceSynchronize();
    }

    // Crea l'immagine cartoonizzata sulla GPU
    create_cartoon_image<<<(num_pixels + 255) / 256, 256>>>(d_pixels, d_assignments, d_centroids, num_pixels);
    hipDeviceSynchronize();

    // Copia i pixel finali sulla CPU
    hipMemcpy(pixels, d_pixels, num_pixels * sizeof(Color), hipMemcpyDeviceToHost);

    // Crea l'immagine finale
    for (int i = 0; i < num_pixels; i++) {
        cv::Vec3b new_color(
            (unsigned char)(pixels[i].b * 255),
            (unsigned char)(pixels[i].g * 255),
            (unsigned char)(pixels[i].r * 255)
        );
        image.at<cv::Vec3b>(i / width, i % width) = new_color;
    }

    // Salva l'immagine cartoonizzata
    cv::imwrite("images/cartoon_image.jpg", image);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    // Calcolo del tempo trascorso
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Tempo di esecuzione %fms\n", milliseconds);

    // Pulizia
    free(pixels);
    free(centroids);
    free(cluster_sizes);
    hipFree(d_pixels);
    hipFree(d_centroids);
    hipFree(d_assignments);
    hipFree(d_cluster_sizes);

    return 0;
}
